#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

__global__
void blurKernel(unsigned char *Pout, unsigned char *Pin, int w, int h, int BLUR_SIZE)
{
	int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
	int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;

	if (colIdx < w && rowIdx < h)
	{
		int pixValR = 0;
		int pixValG = 0;
		int pixValB = 0;
		int pixCounter = 0;
		// Get the average of the surrounding BLUR_SIZE * BLUR_SIZE box.
		for (int blurRowIdxOffset = -BLUR_SIZE; blurRowIdxOffset < BLUR_SIZE + 1; blurRowIdxOffset++)
		{
			for (int blurColIdxOffset = -BLUR_SIZE; blurColIdxOffset < BLUR_SIZE + 1; blurColIdxOffset++)
			{
				int tarColIdx = colIdx + blurColIdxOffset;
				int tarRowIdx = rowIdx + blurRowIdxOffset;
				// Verify we have a valid image pixel
				if (tarColIdx >= 0 && tarColIdx < w && tarRowIdx >= 0 && tarRowIdx < h)
				{
					// pixVal += Pin[tarRowIdx * w + tarColIdx];
					int tarPixIdx = 3 * (tarRowIdx * w + tarColIdx);
					pixValR += Pin[tarPixIdx + 0];
					pixValG += Pin[tarPixIdx + 1];
					pixValB += Pin[tarPixIdx + 2];
					pixCounter++;
				}
			}
		}
		int channelIdx = 3 * (rowIdx * w + colIdx);
		Pout[channelIdx + 0] = (unsigned char)(pixValR / pixCounter);
		Pout[channelIdx + 1] = (unsigned char)(pixValG / pixCounter);
		Pout[channelIdx + 2] = (unsigned char)(pixValB / pixCounter);
		int a = 2;
		// Pout[rowIdx * w + colIdx] = (unsigned char)(pixVal / pixCounter);
	}
}


void Blur(unsigned char* Pout, unsigned char* Pin, int width, int height, int channel)
{
	unsigned char* d_Pin, *d_Pout;
	int size = width * height * channel * sizeof(unsigned char);

	hipMalloc((void**)&d_Pin, size);
	hipMemcpy(d_Pin, Pin, size, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_Pout, size);

	dim3 dimGrid(ceil(width / 16.0), ceil(height / 16.0), 1);
	dim3 dimBlock(16, 16, 1);

	blurKernel << <dimGrid, dimBlock >> > (d_Pout, d_Pin, width, height, 3);

	hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);

	hipFree(d_Pin);
	hipFree(d_Pout);
}

int main()
{

	int w, h, n;
	unsigned char *data = stbi_load("rgba.png", &w, &h, &n, 0);
	unsigned char *oData = new unsigned char[w * h * n];

	Blur(oData, data, w, h, n);

	stbi_write_png("write.png", w, h, n, oData, 0);
	stbi_image_free(data);

    return 0;
}

